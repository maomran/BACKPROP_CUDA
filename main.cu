#include <cstdio>
#include <cstdlib>
#include <ctime>

#include "fclayer.h"
#include "relylayer.h"
#include "sgd.h"
#include "funobj.h"
#include "model.h"
#include "mnist.h"
#include "csv_logger.h"
#include "utils.h"


int main() {
    // Always initialize seed to some random value
    srand(static_cast<unsigned>(time(0)));

    // Prepare events for measuring time on CUDA
    float elapsedTime = 0.0;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // Print our current configuration for this training
    // Configuration::printCurrentConfiguration();
    // Configuration::printCUDAConfiguration();

    // Read both training and test dataset
    MNISTDataSet* trainDataset = new MNISTDataSet(TRAIN);
    MNISTDataSet* testDataset = new MNISTDataSet(TEST);

    // Prepare optimizer and loss function
    float learningRate = LR;
    SGD* optimizer = new SGD(learningRate);
    CrossEntropyLoss* loss = new CrossEntropyLoss();

    // Prepare model
    Model* model = new Model(optimizer, loss);
    model->addLayer(new DenseLayer(28*28, 100));
    model->addLayer(new ReluLayer(100));
    model->addLayer(new DenseLayer(100, 10));

    // Prepare logger that will help us gather timings from experiments
    CSVLogger* logger = new CSVLogger(LOG_FILE_NAME);

    // Run some epochs
    int epochs = EPOCHS;
    int batchSize = BATCHSIZE;
    int numberOfTrainBatches = trainDataset->getSize() / batchSize;
    int numberOfTestBatches = testDataset->getSize() / batchSize;
    for (int e = 0; e < epochs; e++) {
        float trainingLoss = 0.0, trainingAccuracy = 0.0;
        double trainingForwardTime = 0.0, trainingBackwardTime = 0.0;
        printf("Epoch %d:\n", e);
        for (int batch = 0; batch < numberOfTrainBatches; batch++) {
            // Fetch batch from dataset
            Tensor2D* images = trainDataset->getBatchOfImages(batch, batchSize);
            Tensor2D* labels = trainDataset->getBatchOfLabels(batch, batchSize);

            // Forward pass
            hipEventRecord(start, 0);
            Tensor2D* output = model->forward(images);
            hipEventRecord(end, 0);
            hipEventSynchronize(end);

            // Save statistics
            trainingLoss += loss->getLoss(output, labels);
            trainingAccuracy += loss->getAccuracy(output, labels);
            hipEventElapsedTime(&elapsedTime, start, end);
            trainingForwardTime += elapsedTime;

            // Backward pass
            hipEventRecord(start, 0);
            model->backward(output, labels);
            hipEventRecord(end, 0);
            hipEventSynchronize(end);

            // Save statistics
            hipEventElapsedTime(&elapsedTime, start, end);
            trainingBackwardTime += elapsedTime;

            // Clean data for this batch
            delete images;
            delete labels;
        }

        // Calculate mean training metrics
        trainingLoss /= numberOfTrainBatches;
        trainingAccuracy /= numberOfTrainBatches;
        printf("  - [Train] Loss=%.5f\n", trainingLoss);
        printf("  - [Train] Accuracy=%.5f%%\n", trainingAccuracy);
        printf("  - [Train] Total Forward Time=%.5fms\n", trainingForwardTime);
        printf("  - [Train] Total Backward Time=%.5fms\n", trainingBackwardTime);
        printf("  - [Train] Batch Forward Time=%.5fms\n", trainingForwardTime / numberOfTrainBatches);
        printf("  - [Train] Batch Backward Time=%.5fms\n", trainingBackwardTime / numberOfTrainBatches);

        // Check model performance on test set
        float testLoss = 0.0, testAccuracy = 0.0;
        for (int batch = 0; batch < numberOfTestBatches; batch++) {
            // Fetch batch from dataset
            Tensor2D* images = testDataset->getBatchOfImages(batch, batchSize);
            Tensor2D* labels = testDataset->getBatchOfLabels(batch, batchSize);

            // Forward pass
            Tensor2D* output = model->forward(images);

            // Print error
            testLoss += loss->getLoss(output, labels);
            testAccuracy += loss->getAccuracy(output, labels);

            // Clean data for this batch
            delete images;
            delete labels;
        }

        // Calculate mean testing metrics
        testLoss /= numberOfTestBatches;
        testAccuracy /= numberOfTestBatches;
        printf("  - [Test] Loss=%.5f\n", testLoss);
        printf("  - [Test] Accuracy=%.5f%%\n", testAccuracy);
        printf("\n");

        // Save times to the logger
        logger->logEpoch(trainingLoss, trainingAccuracy,
                         testLoss, testAccuracy,
                         trainingForwardTime, trainingBackwardTime,
                         trainingForwardTime / numberOfTrainBatches,
                         trainingBackwardTime / numberOfTrainBatches);

        // Shuffle both datasets before next e!
        trainDataset->shuffle();
        testDataset->shuffle();
    }
    delete logger;
    return 0;
}
