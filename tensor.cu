#include "hip/hip_runtime.h"
#include "tensor.h"
#include "utils.h"

__global__ void tensorAdd(const float *A, const float *B, float *C, int m, int n)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < m && col < n)
    {
        C[row*m + col] = A[row*m + col] + B[row*m + col];
    }
}

__global__ void tensorSub(const float *A, const float *B, float *C, int m, int n)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < m && col < n)
    {
        C[row*m + col] = A[row*m + col] - B[row*m + col];
    }
}


__global__ void tensorScale(float *A, float scale, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < m && col < n){
        A[col*m + row] *= scale;
    }
}

__global__
void tensorMul(float *A, float *B, float *C, int m, int n, int k){
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    float sum = 0;
    if( col < k && row < m) 
    {
        for( i = 0; i < n; i++) 
        {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }

}

tensor::tensor(int row, int col) {
    this->row = row;
    this->col = col;
    if (this->row && this->col) {
        hipMalloc((void **)&(this->d_data), this->row*this->col*sizeof(float));
    } else {
        this->d_data = NULL;
    }
}

tensor::tensor(int row, int col, float** h_data) {
    this->row = row;
    this->col = col;
    if (this->row && this->col) {
        hipMalloc((void **)&(this->d_data), this->row*this->col*sizeof(float));
        hipMemcpy(this->d_data, *h_data, 
            this->row*this->col*sizeof(float), hipMemcpyHostToDevice);
    } 
    else {
        this->d_data = NULL;
    }
}

tensor::tensor(int row, int col, float* d_data) {
    this->row = row;
    this->col = col;
    this->d_data = d_data;
}

tensor::~tensor() {
    hipFree(this->d_data);
}

// int tensor::getSize(tensorAxis axis) {
//     if (axis == X) {
//         return this->row;
//     } else if (axis == Y) {
//         return this->col;
//     }
//     return -1;
// }

float* tensor::DevData() {
    return this->d_data;
}

float** tensor::Dev2Host() {
    float** h_data = new float*[this->col];
    *h_data = new float[this->col * this->row];
    for (int i = 1; i < this->col; i++) 
        h_data[i] = h_data[i-1] + this->row;
    hipMemcpy(*h_data, this->d_data, this->row*this->col*sizeof(float), hipMemcpyDeviceToHost);
    return h_data;
}


tensor* tensor::add(tensor* tensor_t, tensor* output) {
    if (this->row != tensor_t->row || this->col != tensor_t->col) {
        printf("ERROR! Cannot add matrix with size %dx%d to matrix %dx%d.\n",
               tensor_t->row, tensor_t->row, this->row, this->col);
        exit(1);
    }

    dim3 dimBlock(TIDX, TIDY);
    dim3 dimGrid((this->row + dimBlock.x)/dimBlock.x,
                   (this->col + dimBlock.y)/dimBlock.y);
    tensorAdd<<<dimGrid, dimBlock>>>(this->DevData(), tensor_t->DevData(),output->DevData, this->row, this->col);
    return output;
}

tensor* tensor::subtract(tensor* tensor_t) {
    if (this->row != tensor_t->row || this->col != tensor_t->col) {
        printf("ERROR! Cannot sub matrix with size %dx%d to matrix %dx%d.\n",
               tensor_t->row, ten sor_t->row, this->row, this->col);
        exit(1);
    }
    dim3 dimBlock(TIDX, TIDY);
    dim3 dimGrid((this->row + dimBlock.x)/dimBlock.x,
                   (this->col + dimBlock.y)/dimBlock.y);
    tensorSub<<<dimGrid, dimBlock>>>(this->DevData(), tensor_t->DevData(),output->DevData, this->row, this->col);
    return output;
}


void tensor::scale(float factor) {
    dim3 dimBlock(TIDX, TIDY);
    dim3 dimGrid((this->row + dimBlock.x)/dimBlock.x,
                   (this->col + dimBlock.y)/dimBlock.y);
    tensorScale<<<dimGrid, dimBlock>>>(this->DevData(), factor, this->row, this->col);
}


tensor* tensor::multiply(tensor* tensor_t, tensor* output) {
    if (this->row != tensor_t->col) {
        printf("ERROR! Cannot multiply matrices with shape %dx%d and %dx%d.\n",
               this->row, this->col, tensor_t->row, tensor_t->col);
        exit(1);
    }

    dim3 dimBlock(TIDX, TIDY);
    dim3 dimGrid((this->row + dimBlock.x)/dimBlock.x,
                   (this->col + dimBlock.y)/dimBlock.y);
 
        // Defer calculations on GPU
        tensorMul<<<dimGrid, dimBlock>>>(
            this->DevData(), tensor_t->DevData(),output->DevData(),
            this->row, this->col, tensor->col
        );
    return output;
}


Tensor1D* tensor::avg(Tensor1D* output) {
    int dimBlock = TIDX;
    int dimGrid = (this->row + dimBlock)/dimBlock;
    avg<<<dimGrid, dimBlock>>>(this->DevData(), this->row, this->col, output->DevData());
    return output;
}

void tensor::toString() {
    float** values = this->Dev2Host();
    for (int y = 0; y < this->col; y++) {
        for (int x = 0; x < this->row; x++) {
            printf("%8.5f; ", values[y][x]);
        }
        printf("\n");
    }
    delete[] values;
}

